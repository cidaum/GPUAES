#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <sys/time.h>
#include "aes.h"

#define aes_mul(a, b) ((a)&&(b)?iLogTable[(logTable[(a)]+logTable[(b)])%0xff]:0)
#define caes_mul(a, b) ((a)&&(b)?CiLogTable[(ClogTable[(a)]+ClogTable[(b)])%0xff]:0)
#define GET(M,X,Y) ((M)[((Y) << 2) + (X)])

int const THREADS = 512;

__device__ void C2SubBytes(uint8_t *estado) {
		estado[threadIdx.x] = Csbox[estado[threadIdx.x]];
}

__device__ void C2InvSubBytes(uint8_t *estado) {
		estado[threadIdx.x] = CInvSbox[estado[threadIdx.x]];
}

__device__ void C2ShiftRows(uint8_t *estado) {
		unsigned int idx  = threadIdx.x;
		int row = idx % 4;
		uint8_t t;
	
		t = estado[((idx + 4*row) % 16) + ((idx >> 4 ) << 4)];
	
		__syncthreads();

		estado[idx] = t;
}

__device__ void C2InvShiftRows(uint8_t *estado) {
		unsigned int idx  = threadIdx.x;
		int row = idx % 4;
		uint8_t t;
	
		t = estado[((idx - 4*row) % 16) + ((idx >> 4 ) << 4)];
	
		__syncthreads();

		estado[idx] = t;
}

__device__ void C2MixColumns(uint8_t *estado) {
		unsigned int idx = threadIdx.x;
		int base = idx % 4;
		uint8_t t;

		if(base == 0) t = caes_mul(0x02, estado[idx]) ^ caes_mul(0x03, estado[idx+1]) ^ estado[idx+2] ^ estado[idx+3];
		if(base == 1) t = estado[idx-1] ^ caes_mul(0x02, estado[idx]) ^ caes_mul(0x03, estado[idx+1]) ^ estado[idx+2];
		if(base == 2) t = estado[idx-2] ^ estado[idx-1] ^ caes_mul(0x02, estado[idx]) ^ caes_mul(0x03, estado[idx+1]);
		if(base == 3) t = caes_mul(0x03, estado[idx-3]) ^ estado[idx-2] ^ estado[idx-1] ^ caes_mul(0x02, estado[idx]);
	
		__syncthreads();

		estado[idx] = t;
}

__device__ void C2InvMixColumns(uint8_t *estado) {
		unsigned int idx = threadIdx.x;
		int base = idx % 4;
		uint8_t t;

		if(base == 0) t = caes_mul(0x0e, estado[idx]) ^ caes_mul(0x0b, estado[idx+1]) ^ caes_mul(0x0d, estado[idx+2]) ^ caes_mul(0x09, estado[idx+3]);
		if(base == 1) t = caes_mul(0x09, estado[idx-1]) ^ caes_mul(0x0e, estado[idx]) ^ caes_mul(0x0b, estado[idx+1]) ^ caes_mul(0x0d, estado[idx+2]);
		if(base == 2) t = caes_mul(0x0d, estado[idx-2]) ^ caes_mul(0x09, estado[idx-1]) ^ caes_mul(0x0e, estado[idx]) ^ caes_mul(0x0b, estado[idx+1]);
		if(base == 3) t = caes_mul(0x0b, estado[idx-3]) ^ caes_mul(0x0d, estado[idx-2]) ^ caes_mul(0x09, estado[idx-1]) ^ caes_mul(0x0e, estado[idx]);
	
		__syncthreads();

		estado[idx] = t;
}

__device__ void C2AddRoundKey(uint8_t *estado, uint8_t *chave) {
		estado[threadIdx.x] ^= chave[threadIdx.x % 16];
}


//Substitui o estado pelas entradas da S_BOX
__global__ void CSubBytes(uint8_t *estado) {
	estado[(blockIdx.x*blockDim.x)+(blockIdx.y*gridDim.x*blockDim.x)+threadIdx.x] = Csbox[estado[(blockIdx.x*blockDim.x)+(blockIdx.y*gridDim.x*blockDim.x)+threadIdx.x]];
}

void SubBytes(uint8_t *estado, uint64_t offset) {
	for(uint64_t j=0; j<offset; j++) {
		for(register int i=0; i<16; i++){
			estado[i+(16*j)] = Sbox[estado[i+(16*j)]];
		}
	}
}

__global__ void CInvSubBytes(uint8_t *estado) {
		estado[(blockIdx.x*blockDim.x)+(blockIdx.y*gridDim.x*blockDim.x)+threadIdx.x] = CInvSbox[estado[(blockIdx.x*blockDim.x)+(blockIdx.y*gridDim.x*blockDim.x)+threadIdx.x]];
}

void InvSubBytes(uint8_t *estado, uint64_t offset) {
	for(uint64_t j=0; j<offset; j++) {
		for(register int i=0; i<16; i++){
			estado[i+(16*j)]= InvSbox[estado[i+(16*j)]];
		}
	}
}

__global__ void CShiftRows(uint8_t *estado) {
	uint64_t idx  = (blockIdx.x*blockDim.x)+(blockIdx.y*gridDim.x*blockDim.x)+threadIdx.x;
	int row = idx % 4;
	uint8_t t;

	t = estado[((idx + 4*row) % 16) + ((idx >> 4 ) << 4)];

	__syncthreads();

	estado[idx] = t;
}

void ShiftRows(uint8_t *estado, uint64_t offset) {
	for(uint64_t j=0; j<offset; j++) {
		uint8_t t[16];
		for(register int i=0; i<16; i++){
			uint64_t idx  = i+(16*j);
			int row = idx % 4;
			
			t[i] = estado[((idx + 4*row) % 16) + ((idx >> 4) << 4)];
		}
		for(register int i=0; i<16; i++) {
			estado[i+(16*j)] = t[i];
		}
	}
}

__global__ void CInvShiftRows(uint8_t *estado) {
	uint64_t idx  = (blockIdx.x*blockDim.x)+(blockIdx.y*gridDim.x*blockDim.x)+threadIdx.x;
	int row = idx % 4;
	uint8_t t;

	t = estado[((idx - 4*row) % 16) + ((idx >> 4 ) << 4)];

	__syncthreads();

	estado[idx] = t;
}


void InvShiftRows(uint8_t *estado, uint64_t offset) {
	for(uint64_t j=0; j<offset; j++) {
		uint8_t t[16];
		for(register int i=0; i<16; i++){
			uint64_t idx  = i+(16*j);
			int row = idx % 4;
			
			t[i] = estado[((idx - 4*row) % 16) + ((idx >> 4) << 4)];
		}
		for(register int i=0; i<16; i++) {
			estado[i+(16*j)] = t[i];
		}
	}
}

__global__ void CMixColumns(uint8_t *estado) {
	uint64_t idx = (blockIdx.x*blockDim.x)+(blockIdx.y*gridDim.x*blockDim.x)+threadIdx.x;
	uint8_t base = idx % 4;
	uint8_t t;

	if(base == 0) t = caes_mul(0x02, estado[idx]) ^ caes_mul(0x03, estado[idx+1]) ^ estado[idx+2] ^ estado[idx+3];
	if(base == 1) t = estado[idx-1] ^ caes_mul(0x02, estado[idx]) ^ caes_mul(0x03, estado[idx+1]) ^ estado[idx+2];
	if(base == 2) t = estado[idx-2] ^ estado[idx-1] ^ caes_mul(0x02, estado[idx]) ^ caes_mul(0x03, estado[idx+1]);
	if(base == 3) t = caes_mul(0x03, estado[idx-3]) ^ estado[idx-2] ^ estado[idx-1] ^ caes_mul(0x02, estado[idx]);

	__syncthreads();

	estado[idx] = t;
}

void MixColumns(uint8_t *estado, uint64_t offset) {
	for(uint64_t j=0; j<offset; j++) {
		uint8_t t[16];
		for(register int i=0; i<16; i++) {
			uint64_t idx = (i+(16*j));
			uint8_t base = idx % 4;
	
			if(base == 0) t[i] = aes_mul(0x02, estado[idx]) ^ aes_mul(0x03, estado[idx+1]) ^ estado[idx+2] ^ estado[idx+3];
			if(base == 1) t[i] = estado[idx-1] ^ aes_mul(0x02, estado[idx]) ^ aes_mul(0x03, estado[idx+1]) ^ estado[idx+2];
			if(base == 2) t[i] = estado[idx-2] ^ estado[idx-1] ^ aes_mul(0x02, estado[idx]) ^ aes_mul(0x03, estado[idx+1]);
			if(base == 3) t[i] = aes_mul(0x03, estado[idx-3]) ^ estado[idx-2] ^ estado[idx-1] ^ aes_mul(0x02, estado[idx]);
		}
		for(register int i=0; i<16; i++) {
			estado[i+(16*j)] = t[i];
		}
	}
}

__global__ void CInvMixColumns(uint8_t *estado) {
	uint64_t idx = (blockIdx.x*blockDim.x)+(blockIdx.y*gridDim.x*blockDim.x)+threadIdx.x;
	uint8_t base = idx % 4;
	uint8_t t;

	if(base == 0) t = caes_mul(0x0e, estado[idx]) ^ caes_mul(0x0b, estado[idx+1]) ^ caes_mul(0x0d, estado[idx+2]) ^ caes_mul(0x09, estado[idx+3]);
	if(base == 1) t = caes_mul(0x09, estado[idx-1]) ^ caes_mul(0x0e, estado[idx]) ^ caes_mul(0x0b, estado[idx+1]) ^ caes_mul(0x0d, estado[idx+2]);
	if(base == 2) t = caes_mul(0x0d, estado[idx-2]) ^ caes_mul(0x09, estado[idx-1]) ^ caes_mul(0x0e, estado[idx]) ^ caes_mul(0x0b, estado[idx+1]);
	if(base == 3) t = caes_mul(0x0b, estado[idx-3]) ^ caes_mul(0x0d, estado[idx-2]) ^ caes_mul(0x09, estado[idx-1]) ^ caes_mul(0x0e, estado[idx]);
	
	__syncthreads();

	estado[idx] = t;
}

void InvMixColumns(uint8_t *estado, uint64_t offset) {
	for(uint64_t j=0; j<offset; j++) {
		uint8_t t[16];
		for(register int i=0; i<16; i++) {
			uint64_t idx = (i+(16*j));
			uint8_t base = idx % 4;
		
			if(base == 0) t[i] = aes_mul(0x0e, estado[idx]) ^ aes_mul(0x0b, estado[idx+1]) ^ aes_mul(0x0d, estado[idx+2]) ^ aes_mul(0x09, estado[idx+3]);
			if(base == 1) t[i] = aes_mul(0x09, estado[idx-1]) ^ aes_mul(0x0e, estado[idx]) ^ aes_mul(0x0b, estado[idx+1]) ^ aes_mul(0x0d, estado[idx+2]);
			if(base == 2) t[i] = aes_mul(0x0d, estado[idx-2]) ^ aes_mul(0x09, estado[idx-1]) ^ aes_mul(0x0e, estado[idx]) ^ aes_mul(0x0b, estado[idx+1]);
			if(base == 3) t[i] = aes_mul(0x0b, estado[idx-3]) ^ aes_mul(0x0d, estado[idx-2]) ^ aes_mul(0x09, estado[idx-1]) ^ aes_mul(0x0e, estado[idx]);
		}
		for(register int i=0; i<16; i++) {
			estado[i+(16*j)] = t[i];
		}
	}
}

__global__ void CAddRoundKey(uint8_t *estado, uint8_t *chave) {
		estado[(blockIdx.x*blockDim.x)+(blockIdx.y*gridDim.x*blockDim.x)+threadIdx.x] ^= chave[((blockIdx.x*blockDim.x)+(blockIdx.y*gridDim.x*blockDim.x)+threadIdx.x) % 16];
}

void AddRoundKey(uint8_t *estado, uint8_t *chave, uint64_t offset) {
	for(uint64_t j=0; j<offset; j++) {	
		for(uint8_t i=0; i<16; i++) {
			estado[i+(16*j)] ^= chave[i];
		}
	}
}

__global__ void C2InvAes(uint8_t *cp, uint8_t *cW, uint8_t Nr) {
	__shared__ uint8_t estado[THREADS];
	register int i;
	estado[threadIdx.x] = cp[(blockIdx.x*blockDim.x)+(blockIdx.y*blockDim.x*gridDim.x)+threadIdx.x];
	__syncthreads();
	C2AddRoundKey(estado, cW+(Nr << 4));
	for(i=Nr; i>1; i--) {
		C2InvShiftRows(estado);
		C2InvSubBytes(estado);
		C2AddRoundKey(estado, cW+((i-1) << 4));
		C2InvMixColumns(estado);
	}
	C2InvShiftRows(estado);
	C2InvSubBytes(estado);
	C2AddRoundKey(estado, cW);
	__syncthreads();
	cp[(blockIdx.x*blockDim.x)+(blockIdx.y*blockDim.x*gridDim.x)+threadIdx.x] = estado[threadIdx.x];
}

__global__ void C2Aes(uint8_t *cp, uint8_t *cW, uint8_t Nr) {
	__shared__ uint8_t estado[THREADS];
	register int i;
	estado[threadIdx.x] = cp[(blockIdx.x*blockDim.x)+(blockIdx.y*blockDim.x*gridDim.x)+threadIdx.x];
	__syncthreads();
	C2AddRoundKey(estado, cW);
	for(i=1; i<Nr; i++) {
		C2SubBytes(estado);
		C2ShiftRows(estado);
		C2MixColumns(estado);
		C2AddRoundKey(estado, cW+(i << 4));
	}
	C2SubBytes(estado);
	C2ShiftRows(estado);
	C2AddRoundKey(estado, cW+(i << 4));
	__syncthreads();
	cp[(blockIdx.x*blockDim.x)+(blockIdx.y*blockDim.x*gridDim.x)+threadIdx.x] = estado[threadIdx.x];
}

void cinvAes(uint8_t *cp, uint8_t *cW, uint8_t Nr, dim3 numeroBlocos, uint16_t numeroThreads, uint64_t n) {
	
	register uint8_t i;
//	register uint64_t j;
//  	uint8_t tmp[16*n];
//	hipMemcpy(tmp, cp, sizeof(uint8_t)*16*n, hipMemcpyDeviceToHost);
//  	printf("0 str ");
//  	for(j=0; j < 16*n; j++) {
//  		printf("%02X", tmp[j]);
//  	}
//  	printf("\n");
	CAddRoundKey<<<numeroBlocos,numeroThreads>>>(cp, cW+(Nr << 4));
//  	hipMemcpy(tmp, cp, sizeof(uint8_t)*16*n, hipMemcpyDeviceToHost);
//  	printf("0 add ");
//  	for(j=0; j < 16*n; j++) {
//  		printf("%02X", tmp[j]);
//  	}
//  	printf("\n");
	for(i=Nr; i>1; i--) {
		CInvShiftRows<<<numeroBlocos,numeroThreads>>>(cp);
//  		hipMemcpy(tmp, cp, sizeof(uint8_t)*16*n, hipMemcpyDeviceToHost);
//  		printf("%d shi ",i);
//  		for(j=0; j < 16*n; j++) {
//  			printf("%02X", tmp[j]);
//  		}
//  		printf("\n");
		CInvSubBytes<<<numeroBlocos,numeroThreads>>>(cp);
//  		hipMemcpy(tmp, cp, sizeof(uint8_t)*16*n, hipMemcpyDeviceToHost);
//  		printf("%d sub ",i);
// 		for(j=0; j < 16*n; j++) {
//  			printf("%02X", tmp[j]);
//  		}
//  		printf("\n");
		CAddRoundKey<<<numeroBlocos,numeroThreads>>>(cp, cW+((i-1) << 4));
//  		hipMemcpy(tmp, cp, sizeof(uint8_t)*16*n, hipMemcpyDeviceToHost);
// 		printf("%d add ",i);
//  		for(j=0; j < 16*n; j++) {
//  			printf("%02X", tmp[j]);
//  		}
//		printf("\n");
		CInvMixColumns<<<numeroBlocos,numeroThreads>>>(cp);
//  		hipMemcpy(tmp, cp, sizeof(uint8_t)*16*n, hipMemcpyDeviceToHost);
//  		printf("%d mix ",i);
//  		for(j=0; j < 16*n; j++) {
//  			printf("%02X", tmp[j]);
//  		}
//  		printf("\n");
	}
	CInvShiftRows<<<numeroBlocos,numeroThreads>>>(cp);
	CInvSubBytes<<<numeroBlocos,numeroThreads>>>(cp);
	CAddRoundKey<<<numeroBlocos,numeroThreads>>>(cp, cW);
	
}

void caes(uint8_t *cp, uint8_t *cW, uint8_t Nr, dim3 numeroBlocos, uint16_t numeroThreads, uint64_t n) {

	register uint8_t i;
//	register uint64_t j;
//	uint8_t tmp[16*n];
	CAddRoundKey<<<numeroBlocos,numeroThreads>>>(cp, cW);
//	hipMemcpy(tmp, cp, sizeof(uint8_t)*16*n, hipMemcpyDeviceToHost);
//	printf("0 add ");
//	for(j=0; j < 16*n; j++) {
//		printf("%02X", tmp[j]);
//	}
//	printf("\n");
	for(i=1; i<Nr; i++) {
		CSubBytes<<<numeroBlocos,numeroThreads>>>(cp);
//		hipMemcpy(tmp, cp, sizeof(uint8_t)*16*n, hipMemcpyDeviceToHost);
//		printf("%d sub ",i);
//		for(j=0; j < 16*n; j++) {
//			printf("%02X", tmp[j]);
//		}
//		printf("\n");
		CShiftRows<<<numeroBlocos,numeroThreads>>>(cp);
//		hipMemcpy(tmp, cp, sizeof(uint8_t)*16*n, hipMemcpyDeviceToHost);
//		printf("%d shi ",i);
//		for(j=0; j < 16*n; j++) {
//			printf("%02X", tmp[j]);
//		}
//		printf("\n");
		CMixColumns<<<numeroBlocos,numeroThreads>>>(cp);
//		hipMemcpy(tmp, cp, sizeof(uint8_t)*16*n, hipMemcpyDeviceToHost);
//		printf("%d mix ",i);
//		for(j=0; j < 16*n; j++) {
//			printf("%02X", tmp[j]);
//		}
//		printf("\n");
		CAddRoundKey<<<numeroBlocos,numeroThreads>>>(cp, cW+(i << 4));
//		hipMemcpy(tmp, cp, sizeof(uint8_t)*16*n, hipMemcpyDeviceToHost);
//		printf("%d add ",i);
//		for(j=0; j < 16*n; j++) {
//			printf("%02X", tmp[j]);
//		}
//		printf("\n");
	}
	CSubBytes<<<numeroBlocos,numeroThreads>>>(cp);
	CShiftRows<<<numeroBlocos,numeroThreads>>>(cp);
	CAddRoundKey<<<numeroBlocos,numeroThreads>>>(cp, cW+(i << 4));
}

void aes(uint8_t *tp, uint8_t *W, uint8_t Nr, uint64_t n) {

	register uint8_t i;
//	uint64_t k;
//	printf("-1 add ");
//	for(k=0; k < 16*n; k++) {
//		printf("%02X", tp[k]);
//	}
//	printf("\n");
	AddRoundKey(tp, W, n);
//	printf("0 add ");
//	for(k=0; k < 16*n; k++) {
//		printf("%02X", tp[k]);
//	}
//	printf("\n");
	for(i=1; i<Nr; i++){
		SubBytes(tp, n);
//		printf("%d sub ",i);
//		for(k=0; k < 16*n; k++) {
//			printf("%02X", tp[k]);
//		}
//		printf("\n");
		ShiftRows(tp, n);
//		printf("%d shi ",i);
//		for(k=0; k < 16*n; k++) {
//			printf("%02X", tp[k]);
//		}
//		printf("\n");
		MixColumns(tp, n);
//		printf("%d mix ",i);
//		for(k=0; k < 16*n; k++) {
//			printf("%02X", tp[k]);
//		}
//		printf("\n");
		AddRoundKey(tp, W+(i << 4), n);
//		printf("%d add ",i);
//		for(k=0; k < 16*n; k++) {
//			printf("%02X", tp[k]);
//		}
//		printf("\n");
	}
	SubBytes(tp, n);
	ShiftRows(tp, n);
	AddRoundKey(tp, W+(i << 4), n);
//	printf("%d add ",i);
//	for(k=0; k < 16*n; k++) {
//		printf("%02X", tp[k]);
//	}
//	printf("\n");
}

void invAes(uint8_t *tp, uint8_t *W, uint8_t Nr, uint64_t n) {

	register uint8_t i;
//	uint64_t k;
//	printf("-1 add ");
//	for(k=0; k < 16*n; k++) {
//  		printf("%02X", tp[k]);
//	}
//  	printf("\n");
      	AddRoundKey(tp, W+(Nr << 4), n);
//  	printf("0 add ");
//	for(k=0; k < 16*n; k++) {
//  		printf("%02X", tp[k]);
//  	}
//  	printf("\n");
      	for(i=Nr; i>1; i--){
      		InvShiftRows(tp, n);
//  		printf("%d shi ",i);
//  		for(k=0; k < 16*n; k++) {
//  			printf("%02X", tp[k]);
//  		}
//  		printf("\n");
    		InvSubBytes(tp, n);
//  		printf("%d sub ",i);
//  		for(k=0; k < 16*n; k++) {
// 			printf("%02X", tp[k]);
//		}
//		printf("\n");
      		AddRoundKey(tp, W+((i-1) << 4), n);
//		printf("%d add ",i);
//		for(k=0; k < 16*n; k++) {
//			printf("%02X", tp[k]);
//  		}
//  		printf("\n");
      		InvMixColumns(tp, n);
//  		printf("%d mix ",i);
//  		for(k=0; k < 16*n; k++) {
// 			printf("%02X", tp[k]);
//  		}
//  		printf("\n");
	}
	InvShiftRows(tp, n);
	InvSubBytes(tp, n);
	AddRoundKey(tp, W, n);
//	printf("%d add ",i);
//	for(k=0; k < 16*n; k++) {
//		printf("%02X", tp[k]);
//	}
//	printf("\n");
	
}

void ExpandKeys(uint8_t *key, uint8_t keysize, uint8_t *W, uint8_t Nk, uint8_t Nr) {
	uint8_t i, j, cols, temp, tmp[4];
	cols = (Nr + 1) << 2;

	memcpy(W, key, (keysize >> 3)*sizeof(uint8_t));

	for(i=Nk; i<cols; i++) {
		for(j=0; j<4; j++)
			tmp[j] = GET(W, j, i-1);
		if(Nk > 6) {
			if(i % Nk == 0) {
				temp   = Sbox[tmp[0]] ^  (Rcon[i/Nk] & 0x000000ff);
				tmp[0] = Sbox[tmp[1]] ^ ((Rcon[i/Nk] & 0xff000000) >> 24);
				tmp[1] = Sbox[tmp[2]] ^ ((Rcon[i/Nk] & 0x00ff0000) >> 16);
				tmp[2] = Sbox[tmp[3]] ^ ((Rcon[i/Nk] & 0x0000ff00) >>  8);
				tmp[3] = temp;
			} else if(i % Nk == 4) {
				tmp[0] = Sbox[tmp[0]];
				tmp[1] = Sbox[tmp[1]];
				tmp[2] = Sbox[tmp[2]];
				tmp[3] = Sbox[tmp[3]];
			}
		} else {
			if(i % Nk == 0) {
				temp   = Sbox[tmp[0]] ^  (Rcon[i/Nk] & 0x000000ff);
				tmp[0] = Sbox[tmp[1]] ^ ((Rcon[i/Nk] & 0xff000000) >> 24);
				tmp[1] = Sbox[tmp[2]] ^ ((Rcon[i/Nk] & 0x00ff0000) >> 16);
				tmp[2] = Sbox[tmp[3]] ^ ((Rcon[i/Nk] & 0x0000ff00) >>  8);
				tmp[3] = temp;
			}
		}
		for(j=0; j<4; j++)
			GET(W, j, i) = GET(W, j, i-Nk) ^ tmp[j];
	}
}

void aes_serial(uint8_t *in, uint8_t *chave, uint8_t *out, uint8_t tamanhoChave, uint64_t offset, uint8_t acao) {
	uint8_t *W, Nk, Nr;
	Nk = tamanhoChave >> 5;
	Nr = Nk + 6;
	uint64_t size = 4*4*offset*sizeof(uint8_t);
	uint64_t s = ((Nr+1) * sizeof(uint8_t)) << 4;
	W = (uint8_t *)malloc(s);
	ExpandKeys(chave, tamanhoChave, W, Nk, Nr);
	memcpy(out, in, size);
	if(acao) {
		aes(out, W, Nr, offset);
	} else {
		invAes(out, W, Nr, offset);
	}
	//printHexArray(out,sizeof(out));
	//for(register uint8_t i=0; i<(size/sizeof(uint8_t)); i++) {
	//	printf("%d:", out[i]);
	//}
	//printf("\n");
}

void aes_cuda(uint8_t *in, uint8_t *chave, uint8_t *out, uint8_t tamanhoChave, uint64_t offset, dim3 numeroBlocos, uint16_t numeroThreads, uint8_t acao) {
	uint8_t *cp, *W, *cW, Nk, Nr;
	Nk = tamanhoChave >> 5;
	Nr = Nk + 6;
	long size = 4*4*offset*sizeof(uint8_t);
	uint64_t s = ((Nr+1) * sizeof(uint8_t)) << 4;
	W = (uint8_t *)malloc(s);
	hipMalloc((void**)&cW, s);
	ExpandKeys(chave, tamanhoChave, W, Nk, Nr);
	hipMemcpyAsync(cW, W, s, hipMemcpyHostToDevice);
	hipMalloc((void**)&cp, size);
	hipMemcpyAsync(cp, in, size, hipMemcpyHostToDevice);
	if(acao) {
		caes(cp, cW, Nr, numeroBlocos, numeroThreads, offset);
	} else {
		cinvAes(cp, cW, Nr, numeroBlocos, numeroThreads, offset);
	}
	hipMemcpy(out, cp, size, hipMemcpyDeviceToHost);
	hipFree(&cW);
	hipFree(&cp);
	//printHexArray(out,(size/sizeof(uint8_t)));
	//for(register uint8_t i=0; i<(size/sizeof(uint8_t)); i++) {
	//	printf("%d:", out[i]);
	//}
	//printf("\n");
}

void aes_cuda2(uint8_t *in, uint8_t *chave, uint8_t *out, uint8_t tamanhoChave, uint64_t offset, dim3 numeroBlocos, uint8_t acao) {
	uint8_t *cp, *W, *cW, Nk, Nr;
	Nk = tamanhoChave >> 5;
	Nr = Nk + 6;
	long size = 4*4*offset*sizeof(uint8_t);
	uint64_t s = ((Nr+1) * sizeof(uint8_t)) << 4;
	W = (uint8_t *)malloc(s);
	hipMalloc((void**)&cW, s);
	ExpandKeys(chave, tamanhoChave, W, Nk, Nr);
	hipMemcpyAsync(cW, W, s, hipMemcpyHostToDevice);
	hipMalloc((void**)&cp, size);
	hipMemcpyAsync(cp, in, size, hipMemcpyHostToDevice);
	if(acao) {
		C2Aes<<<numeroBlocos, THREADS>>>(cp, cW, Nr);
	} else {
		C2InvAes<<<numeroBlocos, THREADS>>>(cp, cW, Nr);
	}
	hipMemcpy(out, cp, size, hipMemcpyDeviceToHost);
	hipFree(&cW);
	hipFree(&cp);
}

//Transforma a entrada em um array de char
uint8_t stringToByteArray(char *str, uint8_t *array[]) {
	register uint8_t i;
	uint8_t len  = strlen(str) >> 1;
	*array = (uint8_t *)malloc(len * sizeof(uint8_t));

	for(i=0; i<len; i++)
		sscanf(str + i*2, "%02X", *array+i);

	return len;
}

//Imprime a saída em hexa TODO gravar em um arquivo.
void printHexArray(uint8_t *array, uint64_t size) {
	register uint8_t i;
	for(i=0; i<size; i++)
		printf("%02X", array[i]);
	printf("\n");
}

//Popula uma entrada aleaória
void aleatorio(uint8_t *entrada, uint64_t size) {
	for(uint64_t i = 0; i < size; i++)
		entrada[i] = (uint8_t)(rand() % 0xff);
}

//calcula diferença de tempo
double time_diff(struct timeval * prior, struct timeval * latter) {
  double x =
   (double)(latter->tv_usec - prior->tv_usec) / 1000.0L +
   (double)(latter->tv_sec - prior->tv_sec) * 1000.0L;
  return x;
}

int main(int argc, char **argv){
	struct timeval inicio, fim, inicioc, fimc, inicioc2, fimc2, inicios, fims;
	gettimeofday(&inicio,NULL);
	double tempo, totalc, totalc2, totals;
	uint8_t *chave, *outs, *outc, *outc2, *in;
	uint64_t blocos;

        if(argc < 4) {
                printf("Número de parâmetros errados\nUse: aes BLOCOS THREADSPORBLOCO TAMANHOCHAVE TAMANHOENTRADA\n");
		return 1;
        }
	
	dim3 numeroBlocos(atoi(argv[1]), atoi(argv[2]));
	printf("\n x %d y %d z %d \n", numeroBlocos.x, numeroBlocos.y, numeroBlocos.z);
	int numeroThreads = atoi(argv[3]);
	uint8_t tamanhoChave = atoi(argv[4]);
	uint64_t tamanhoIn = atoi(argv[5]);
	
        if(tamanhoChave != 16 && tamanhoChave != 24 && tamanhoChave != 32) {
                printf("Tamanho da chave inválido\n");
                return 1;
        }
	if(tamanhoIn == 0) {
		char *chavein = "000102030405060708090a0b0c0d0e0f";
		char *inin = "3243f6a8885a308d313198a2e037073400112233445566778899aabbccddeeff";	
	        tamanhoChave = stringToByteArray(chavein, &chave);
	        tamanhoIn  = stringToByteArray(inin, &in);
	} else {
	       if(tamanhoIn % 16 != 0) {
			printf("Tamanho de bloco inválido\n Deve ser múltiplo de 16\n resto: %d \n", (tamanhoIn % 16));
			return 1;
	        } else {
			srand(time(NULL));
			chave = (uint8_t *)malloc(tamanhoChave * sizeof(uint8_t));
			in = (uint8_t *)malloc(tamanhoIn * sizeof(uint8_t));
			aleatorio(chave, tamanhoChave);
			aleatorio(in, tamanhoIn);
		}
	}		
	blocos = tamanhoIn / 16;
	printf("%d\n", tamanhoIn);
	printf("Entrada : ");
	printHexArray(in, 32);
	printf("Chave : ");
	printHexArray(chave, tamanhoChave);
	outs = (uint8_t *)malloc(tamanhoIn * sizeof(uint8_t));
	memset(outs, 0, tamanhoIn);
	outc = (uint8_t *)malloc(tamanhoIn * sizeof(uint8_t));
	memset(outc, 0, tamanhoIn);
	outc2 = (uint8_t *)malloc(tamanhoIn * sizeof(uint8_t));
	memset(outc2, 0, tamanhoIn);
	gettimeofday(&fim, NULL);
	tempo = time_diff(&inicio, &fim);
	printf("Tempo de inicialização em ms %f\n",  tempo); 
	
	printf("Criptografa CUDA\n");
	gettimeofday(&inicioc, NULL);
	aes_cuda(in, chave, outc, tamanhoChave << 3, blocos, numeroBlocos, numeroThreads, 1);
	gettimeofday(&fimc, NULL);
	totalc = tempo = time_diff(&inicioc, &fimc);
	printf("Tempo em ms %f\n",  tempo); 
//	printHexArray(outc, 32);

	printf("Criptografa CUDA Otimizado\n");
	gettimeofday(&inicioc2, NULL);
	aes_cuda2(in, chave, outc2, tamanhoChave << 3, blocos, numeroBlocos, 1);
	gettimeofday(&fimc2, NULL);
	totalc2 = tempo = time_diff(&inicioc2, &fimc2);
	printf("Tempo em ms %f\n",  tempo);
//	printHexArray(outc2, 32);

	printf("Criptografa Serial\n");
	gettimeofday(&inicios, NULL);
	aes_serial(in, chave, outs, tamanhoChave << 3, blocos, 1);
	gettimeofday(&fims, NULL);
	totals = tempo = time_diff(&inicios, &fims);
	printf("Tempo em ms %f\n",  tempo); 
//	printHexArray(outs, 32);
	
	printf("Verificando consistencia entre CUDA e Serial: ");
	!memcmp(outs, outc, tamanhoIn)?printf("OK\n"):printf("Falha. Verifique o algoritmo\n");
	printf("Verificando consistencia entre CUDA Otimizado e Serial: ");
	!memcmp(outs, outc2, tamanhoIn)?printf("OK\n"):printf("Falha. Verifique o algoritmo\n");

	printf("Descriptografa CUDA\n");
	gettimeofday(&inicioc, NULL);
	aes_cuda(outc, chave, outc, tamanhoChave << 3, blocos, numeroBlocos, numeroThreads, 0);
	gettimeofday(&fimc, NULL);
	tempo = time_diff(&inicioc, &fimc);
	totalc += tempo;
	printf("Tempo em ms %f\n",  tempo); 
//	printHexArray(outc, 32);
	printf("Verificando algoritmo CUDA: ");
	!memcmp(in, outc, tamanhoIn)?printf("OK\n"):printf("Falha. Verifique o algoritmo\n");

	printf("Descriptografa CUDA Otimizado\n");
	gettimeofday(&inicioc2, NULL);
	aes_cuda2(outc2, chave, outc2, tamanhoChave << 3, blocos, numeroBlocos, 0);
	gettimeofday(&fimc2, NULL);
	tempo = time_diff(&inicioc2, &fimc2);
	totalc2 += tempo;
	printf("Tempo em ms %f\n",  tempo); 
//	printHexArray(outc2, 32);
	printf("Verificando algoritmo CUDA Otimizado: ");
	!memcmp(in, outc2, tamanhoIn)?printf("OK\n"):printf("Falha. Verifique o algoritmo\n");
	
	printf("Descriptografa Serial\n");
	gettimeofday(&inicios, NULL);
	aes_serial(outs, chave, outs, tamanhoChave << 3, blocos, 0);
	gettimeofday(&fims, NULL);
	tempo = time_diff(&inicios, &fims);
	totals += tempo;
	printf("Tempo em ms %f\n",  tempo);
//	printHexArray(outs, 32);
	printf("Verificando algoritmo Serial: ");
	!memcmp(in, outs, tamanhoIn)?printf("OK\n"):printf("Falha. Verifique o algoritmo\n");
	printf("\n");

	printf("Tempo total cuda: %f Tempo total cuda o: %f Tempo total serial: %f", totalc, totalc2, totals); 

        return EXIT_SUCCESS;
}
